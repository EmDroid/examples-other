#include "hip/hip_runtime.h"

#include "MatrixCUDA.hpp"


///@todo Try to utilize thread shared memory for faster memory access.
//       (like it is done in the NVidia SDK example)


// Currently, CUDA requires all textures to be file-scoped.
texture<float, 2, hipReadModeElementType> texA;
texture<float, 2, hipReadModeElementType> texB;

// Access data using NVidia texture units for faster access to global memory.
///@todo Check using of 2-D textures for even faster access.
__global__ void mulCUDArun(
    float * const x,
    const int n,
    const int start, const int end,
    const int blockStart)
{
	int row = (blockIdx.x * blockDim.x) + threadIdx.x;
	int col = (blockIdx.y * blockDim.y) + threadIdx.y;
    if ((row < n) && (col < n)) {
        const size_t rowSize = row * n;
        float * const ptrX = x + rowSize + col;
        // Compute the result[row, col].
        float sum = (0 == start) ? 0.0f : *ptrX;
        for (int i = start; i < end; ++i) {
//            sum = tex2D(texA, 0, 1);
            sum += tex2D(texA, i, row) * tex2D(texB, col, i);
        }
        *ptrX = sum;
    }
}


class CudaException: public Exception {

public:

    CudaException(const char * const name, const hipError_t error)
        : m_name(name)
        , m_error(error)
    {}

protected:

    virtual void onLog(FILE * const logFile) const
    {
        fprintf(logFile, "%s() failed with error %d:\t%s",
            m_name, m_error, ::hipGetErrorString(m_error));
    }

private:

    const char * const m_name;
    const hipError_t m_error;

};

class CuException: public Exception {

public:

    CuException(const char * const name, const hipError_t error)
        : m_name(name)
        , m_error(error)
    {}

protected:

    virtual void onLog(FILE * const logFile) const
    {
        fprintf(logFile, "%s() failed with error %d",
            m_name, m_error /*, ::hipDrvGetErrorString(m_error)*/);
    }

private:

    const char * const m_name;
    const hipError_t m_error;

};

void cudaSafeCall(const char * const funcName, const hipError_t error) {
    if (::hipSuccess != error) {
        throw CudaException(funcName, error);
	}
}

void cuSafeCall(const char * const funcName, const hipError_t error) {
    if (::hipSuccess != error) {
        throw CuException(funcName, error);
	}
}

class CudaMemory {

public:

    CudaMemory(void * data = NULL)
        : m_data(data)
    {}

    ~CudaMemory()
    {
        clear();
    }

public:

    CudaMemory & operator = (hipArray * data)
    {
        if (data != m_data) {
            clear();
            m_data = data;
        }
        return *this;
    }

private:

    void clear()
    {
        if (m_data) {
            ::hipFree(m_data);
        }
    }

private:

    void * m_data;

}; // class CudaMemory


class CudaMemoryArray {

public:

    CudaMemoryArray(hipArray * data = NULL)
        : m_data(data)
    {}

    ~CudaMemoryArray()
    {
        clear();
    }

public:

    CudaMemoryArray & operator = (hipArray * data)
    {
        if (data != m_data) {
            clear();
            m_data = data;
        }
        return *this;
    }

private:

    void clear()
    {
        if (m_data) {
            ::hipFreeArray(m_data);
        }
    }

private:

    hipArray * m_data;

}; // class CudaMemoryArray


void mulCUDA(
    const float * const a,
    const float * const b,
    float * const x,
    const int n,
    const size_t maxComputePerf)
{
    // Prepare the data for CUDA.
    // It is done in advance, to not include this preparation in the time
    // measurement.
    hipArray * gpuInput[2];
    float * gpuOutput;
    const size_t memSize = n * sizeof(float);
    const size_t fullSize = n * memSize;
    hipChannelFormatDesc floatTex = ::hipCreateChannelDesc<float>();
    CudaMemoryArray guardsInput[2];
    for (int i = 0; i < 2; ++i) {
        cudaSafeCall("hipMallocArray",
            ::hipMallocArray(gpuInput + i, &floatTex, n, n));
        guardsInput[i] = gpuInput[i];
    }
    cudaSafeCall("hipMalloc",
        ::hipMalloc(reinterpret_cast< void ** >(&gpuOutput), fullSize));
    CudaMemory guardOutput(gpuOutput);
    // Copy the source data to device,
    cudaSafeCall("hipMemcpy2DToArray", ::hipMemcpy2DToArray(gpuInput[0], 0, 0,
        a, memSize, memSize, n, hipMemcpyHostToDevice));
    cudaSafeCall("hipMemcpy2DToArray", ::hipMemcpy2DToArray(gpuInput[1], 0, 0,
        b, memSize, memSize, n, hipMemcpyHostToDevice));
    // Setup the texture system.
    cudaSafeCall("hipBindTextureToArray", ::hipBindTextureToArray(texA, gpuInput[0], floatTex));
    cudaSafeCall("hipBindTextureToArray", ::hipBindTextureToArray(texB, gpuInput[1], floatTex));//
    // Compute the multiplication of matrices.
    int devId;
    cudaSafeCall("hipGetDevice", ::hipGetDevice(&devId));
    hipDeviceProp_t props;
    cudaSafeCall("hipGetDeviceProperties", ::hipGetDeviceProperties(&props, devId));
    // Determine the maximum thread dimensions.
    int threadDim[2] = {1, 1};
    int threads = 1;
    int dimId = 0;
    while (threadDim[1] < n) {
        threads <<= 1;
        if (threads > props.maxThreadsPerBlock) {
            break;
        }
        threadDim[dimId] <<= 1;
        dimId = ~dimId & 1;
    }
	dim3 dimBlock(threadDim[0], threadDim[1]);
    int blockDim[2] = {1, 1};
    for (dimId = 0; dimId < 2; ++dimId) {
        int threads = threadDim[dimId];
        while (threads < n) {
            threads <<= 1;
            blockDim[dimId] <<= 1;
            if (blockDim[dimId] > props.maxThreadsDim[dimId]) {
                throw Exception("Matrix too big.");
            }
        }
    }
	dim3 dimGrid(blockDim[0], blockDim[1]);
    ///@todo Base the estimation on the GPU props.
    const int nMax = static_cast< size_t >(max(maxComputePerf * 1000.0 / n / n, 1.0));
    //Run the calculation.
    int done = 0;
    // Flush all operations.
    cudaSafeCall("hipDeviceSynchronize", ::hipDeviceSynchronize());
    while (done < n) {
        const int end = min(done + nMax, n);
        mulCUDArun<<<dimGrid, dimBlock>>>(gpuOutput, n, done, end, done * n);
        done = end;
        // Wait for all threads to complete.
	    cudaSafeCall("hipDeviceSynchronize", ::hipDeviceSynchronize());
    }
    // Get the result data from device.
    cudaSafeCall("hipMemcpy", ::hipMemcpy(x, gpuOutput, fullSize, hipMemcpyDeviceToHost));
    // Flush all operations.
    cudaSafeCall("hipDeviceSynchronize", ::hipDeviceSynchronize());
    // Unbind textures.
    cudaSafeCall("hipUnbindTexture", ::hipUnbindTexture(texA));
    cudaSafeCall("hipUnbindTexture", ::hipUnbindTexture(texB));
}
